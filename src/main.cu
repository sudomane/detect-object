#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image/stb_image.h"
#include "stb_image/stb_image_write.h"

#include "image.hpp"

__global__ void my_kernel()
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    printf("Thread n %d\n", i);
}

int main()
{
    int width, height, channels;

    u_char* img = stbi_load("../image.jpeg", &width, &height, &channels, 0);

    if (img == nullptr) return -1;

    printf("Loaded image %dx%dx%d\n", width, height, channels);

    u_char* img_gray = to_grayscale_CPU(img, width, height);

    if (img_gray == nullptr) return -1;

    stbi_write_jpg("../out.jpeg", width, height, 1, img_gray, width);

    free(img);
    free(img_gray);

    return 0;
}