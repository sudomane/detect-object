#include "hip/hip_runtime.h"
#include "image.hpp"

#include <stdio.h>
#include <stdlib.h>

/* CPU API */

namespace CPU
{
void to_grayscale(const u_char* src, u_char* dst, int width, int height, int n_channels)
{
    if (n_channels < 3) return;
    
    for (int i = 0; i < width; i++)
    {
        for (int j = 0; j < height; j++)
        {
            u_char R = src[(j * width + i) * n_channels];
            u_char G = src[(j * width + i) * n_channels + 1];
            u_char B = src[(j * width + i) * n_channels + 2];
            
            dst[j * width + i] = (R + G + B) / 3.f;
        }
    }
}

void conv_2D(const u_char* src, u_char* dst, int width, int height)
{
    float filter[9] = { 16, 8, 16,
                        8,  4, 8,
                        16, 8, 16};

    float top_left, top, top_right; // x x x
    float mid_left, mid, mid_right; // x o x
    float bot_left, bot, bot_right; // x x x

    for (int i = 1; i < width-1; i++)
    {
        for (int j = 1; j < height-1; j++)
        {
            top_left   = src[(j-1) * width + (i-1)]   / filter[0];
            top        = src[(j-1) * width + i]       / filter[1];
            top_right  = src[(j-1) * width + (i+1)]   / filter[2];

            mid_left   = src[j     * width + (i-1)]   / filter[3];
            mid        = src[j     * width + i]       / filter[4];
            mid_right  = src[j     * width + (i+1)]   / filter[5];

            bot_left   = src[(j+1) * width + (i-1)]   / filter[6];
            bot        = src[(j+1) * width + i]       / filter[7];
            bot_right  = src[(j+1) * width + (i+1)]   / filter[8];

            dst[j * width + i] =(u_char) (top_left + top + top_right + 
                                          mid_left + mid + mid_right +
                                          bot_left + bot + bot_right);
        }
    }
}
}; // namespace CPU

/* GPU API */

namespace GPU
{
__global__ void to_grayscale(const u_char* src, u_char* dst, int width, int height,
                             size_t spitch, size_t dpitch, int n_channels)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height || n_channels < 3)
        return;

    const u_char* src_ptr = src + y * spitch;
    u_char*       dst_ptr = dst + y * dpitch;

    dst_ptr[x] = (src_ptr[x * n_channels]       // R
                + src_ptr[x * n_channels + 1]   // G
                + src_ptr[x * n_channels + 2])  // B
                / 3.f; 
}

__global__ void conv_2D(const u_char* src, u_char* dst, int width, int height, size_t pitch)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height) return;


}
}; // namespace GPU