#include "hip/hip_runtime.h"
#include <map>
#include <algorithm>
#include <vector>
#include <set>

#include "img_operations.hpp"

namespace CPU
{
    // Thresholding + Two pass algorithm from wikipedia
    void connected_components(u_char* buffer, int width, int height, int threshold)
    {
        // FIRST PASS
        int index;
        u_char current_label = 1;
        // Add "padding" map {0, 0}, and initial {1, 1} matching for ease of manipulation
        std::map<int, int> label_matching{{0,0}, {1,1}};
        std::array<u_char, 4> neighbors{};
        for (auto line = 0; line < height; line++) {
            for (auto column = 0; column < width; column++) {
                index = line * width + column;
                if (buffer[index] < threshold){
                    buffer[index] = 0;
                    continue; // Background
                }
                std::fill(neighbors.begin(), neighbors.end(), UCHAR_MAX);
                // Check 4 neighbours around + already processed
                if (line > 0 && buffer[index - width]){
                    neighbors[0] = buffer[index - width]; // North pixel
                    if (column > 0 && buffer[index - width - 1]) {
                        neighbors[1] = buffer[index - width - 1]; // North-west pixel
                    }
                    if (column < width - 1 && buffer[index - width + 1]) {
                        neighbors[2] = buffer[index - width + 1]; // North-east pixel
                    }
                }
                if (column > 0 && buffer[index - 1]) {
                    neighbors[3] = buffer[index - 1]; // West pixel
                }
                // Compute minimum neighbour label
                u_char min_label = *std::min_element(neighbors.begin(), neighbors.end());
                // No labelled neighbour
                if (min_label == UCHAR_MAX) {
                    buffer[index] = current_label;
                    current_label++;
                    // Add new label matching and value to the matching map
                    label_matching.insert({current_label, current_label});
                } else {
                    // Update correspondance
                    for (const auto& value : neighbors) {
                        if (value != UCHAR_MAX) {
                            // Update set with the lowest label
                            label_matching.find(value)->second = min_label;
                        }
                    }
                    // Labellise the pixel
                    buffer[index] = min_label;
                }
            }
        }
        // Handle nested label value by going from first to last label and updating corresponding label
        for (auto& match : label_matching) {
            match.second = label_matching.find(match.second)->second;
        }
        // SECOND PASS
        for (auto line = 0; line < height; line++) {
            for (auto column = 0; column < width; column++) {
                index = line * width + column;
                if (buffer[index] != 0) {
                    buffer[index] = label_matching.find(buffer[index])->second * 16; // ADD VALUE FOR DEBUGGING
                }
            }
        }
    }
} // namespace CPU

namespace GPU
{
__global__ void connected_components(u_char* buffer, int width, int height)
{}
} // namespace GPU