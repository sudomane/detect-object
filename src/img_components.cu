#include "hip/hip_runtime.h"
#include <map>
#include <algorithm>
#include <vector>

#include "img_operations.hpp"

namespace CPU
{
    // Thresholding + Two pass algorithm from wikipedia
    void connected_components(u_char* buffer, int width, int height)
    {
        // FIRST PASS
        int index;
        u_char current_label = 2;
        std::map<u_char, u_char> label_matching{{0,0}, {1, 0}}; // Add "padding" sets for ease of manipulation
        // FIXME Problème de propagation des labels avec le set, il faudrait un update "récurrent" mais pas trouvé de solution
        // FIXME propre en utilisant les containers standards... Set serait utile (plutôt que map) mais chiant d'accéder aux key
        // TODO Remplacer map par vector<set<int>> + recherche en vector[].upper_bound(0) ?
        std::array<u_char, 4> neighbors{};
        for (auto line = 0; line < height; line++) {
            for (auto column = 0; column < width; column++) {
                index = line * width + column;
                std::fill(neighbors.begin(), neighbors.end(), UCHAR_MAX);
                if (buffer[index] == 0){
                    continue; // Background
                }
                // Check 4 neighbours around + already processed
                if (line > 0 && buffer[index - width]){
                    neighbors[0] = buffer[index - width]; // North pixel
                    if (column > 0 && buffer[index - width - 1]) {
                        neighbors[1] = buffer[index - width - 1]; // North-west pixel
                    }
                    if (column < width - 1 && buffer[index - width + 1]) {
                        neighbors[2] = buffer[index - width + 1]; // North-east pixel
                    }
                }
                if (column > 0 && buffer[index - 1]) {
                    neighbors[3] = buffer[index - 1]; // West pixel
                }
                // Compute minimum neighbour label
                u_char min_label = *std::min_element(neighbors.begin(), neighbors.end());
                // No labelled neighbour
                if (min_label == UCHAR_MAX) {
                    buffer[index] = current_label;
                    current_label++;
                    // Add new label matching
                    // Add new label value to the matching
                    label_matching.insert({current_label, current_label});
                } else {
                    // Update correspondance
                    for (const auto& value : neighbors) {
                        if (value != UCHAR_MAX) {
                            // Update map with the lowest label
                            label_matching.find(value)->second = min_label;
                        }
                    }
                    // Labellise the pixel
                    buffer[index] = min_label;
                }
            }
        }

        // SECOND PASS
        for (auto line = 0; line < height; line++) {
            for (auto column = 0; column < width; column++) {
                index = line * width + column;
                if (buffer[index] != 0) {
                    buffer[index] = label_matching.find(buffer[index])->second * 16; // ADD VALUE FOR DEBUGGING
                }
            }
        }
    }

    void threshold(u_char* buffer, int width, int height, int threshold) {
        int index;
        for (auto line = 0; line < height; line++) {
            for (auto column = 0; column < width; column++) {
                index = line * width + column;
                if (buffer[index] > threshold) {
                    buffer[index] = 1;
                } else {
                    buffer[index] = 0;
                }
            }
        }
    }
} // namespace CPU

namespace GPU
{
__global__ void connected_components(u_char* buffer, int width, int height)
{}
} // namespace GPU